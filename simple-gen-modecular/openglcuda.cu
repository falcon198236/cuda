
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include ""
#include "openglcuda.cuh"
#include <time.h>

int numElementsRand = 10, numElementsMat = 100, numElementsBestCost = 100;
int sizeRand = numElementsMat * sizeof(int);
int sizeMat = numElementsMat * sizeof(int);
int sizeBestCost = numElementsBestCost * sizeof(int);

