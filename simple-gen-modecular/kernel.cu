#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

//////////////////////////////////////////////////////////////////////////
// OpenGL Graphics includes
#include <helper_gl.h>
#if defined (__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#include <GLUT/glut.h>
#ifndef glutCloseFunc
#define glutCloseFunc glutWMCloseFunc
#endif
#else
#include <GL/freeglut.h>
#endif

#include <stdio.h>
#include "ModecularSurfaceGenerator.h"
/*
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main(int argc, char** argv)
{
	loadPdbfile("E:/WORK/2019_08_24/PDB_files/1IZH.pdb");
	calcuate_model();
	opengl_run(argc, argv);


    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
*/

#include "openglcuda.cuh"

//////////////////////////////////////////////////////////////////////////
#include "CommonPara.h"
#include "ParsePDB.h"
#include "ModecularSurface.h"
#include <time.h>




//////////////////////////////////////////////////////////////////////////
#define WIDTH	800
#define HEIGHT	600
int screen_width = WIDTH;
int screen_height = HEIGHT;
enum ButtonState {
	UP = 1,
	DOWN
};

// State of mouse wheel
enum WheelState {
	WHEEL_UP = 3,
	WHEEL_DOWN = 4
};

ParsePDB pp;
ModecularSurface msf;


static int g_x = 0;
static int g_y = 0;
static int g_z = -80;

static float g_xDiff = 0.0f;
static float g_yDiff = 0.0f;

// mouse state
static int g_mouseState = UP;
// the minimized x/y/z coordinate of all grid
static double g_gridMinXcoord = 0.0f;
static double g_gridMinYcoord = 0.0f;
static double g_gridMinZcoord = 0.0f;
// the maximized x/y/z coordinate of all grid
static double g_gridMaxXcoord = 0.0f;
static double g_gridMaxYcoord = 0.0f;
static double g_gridMaxZcoord = 0.0f;

static int g_numberOfGrid = 60;
double g_iosSurfaceValue = 1.4;


int g_nsurfMode = 2; //1-MC 2-VCMC
int g_ncompute_mode = 4; //1-VWS 2-SAS 3-MS 4-SES
int g_ncolor = 2;//1-pure 2-atom 3-chain
int g_ninout = 1;//1-in and out 2-out 3-in
double g_dblradius = 0.2;//probe radius
double g_dblscale = 2.00;//scale factor

// whether to show atoms
static bool g_showAtoms = false;
// whether to show points
static bool g_showPoints = false;
// whether to show bounding box
static bool g_showBoundingBox = true;
// whether to show triangles
static bool g_showTriangles = true;
// whether to show cuboids
static bool g_showCuboids = false;
// whether to show surfaces
static bool g_showSurface = false;
// whether to show light
static bool g_showLight = false;

static GLfloat g_rotate[] = { 0.0, 0.0, 0.0 }; // Rotation (X,Y,Z)

#define PI       3.14159265358979323846   // pi
//////////////////////////////////////////////////////////////////////////

void drawAtom();
void find_pdbfile(char* pdbfile) {

}
void drawSamplePoints()
{
	glPolygonMode(GL_FRONT_AND_BACK, GL_POINT);
	glPointSize(3);
	glColor3f(0.0f, 1.0f, 0.0f);

}

// Function that handles the drawing of bounding box
void drawCube(int cubeIndex, GLfloat minX, GLfloat maxX, GLfloat minY, GLfloat maxY, GLfloat minZ, GLfloat maxZ, GLfloat r, GLfloat g, GLfloat b)
{
	glBegin(GL_LINE_LOOP);

	//glColor3f(r, g, b);

	if (cubeIndex & (1 << 0))
		glColor3f(r, g, b);
	else
		glColor3f(1.0, 0.0, 0.0);
	glVertex3f(minX, minY, minZ);	// 0

	if (cubeIndex & (1 << 1))
		glColor3f(r, g, b);
	else
		glColor3f(1.0, 0.0, 0.0);
	glVertex3f(maxX, minY, minZ);	// 1

	if (cubeIndex & (1 << 2))
		glColor3f(r, g, b);
	else
		glColor3f(1.0, 0.0, 0.0);
	glVertex3f(maxX, maxY, minZ);	// 2

	if (cubeIndex & (1 << 3))
		glColor3f(r, g, b);
	else
		glColor3f(1.0, 0.0, 0.0);
	glVertex3f(minX, maxY, minZ);	// 3

	if (cubeIndex & (1 << 0))
		glColor3f(r, g, b);
	else
		glColor3f(1.0, 0.0, 0.0);
	glVertex3f(minX, minY, minZ);	// 0

	if (cubeIndex & (1 << 4))
		glColor3f(r, g, b);
	else
		glColor3f(1.0, 0.0, 0.0);
	glVertex3f(minX, minY, maxZ);	// 4

	if (cubeIndex & (1 << 7))
		glColor3f(r, g, b);
	else
		glColor3f(1.0, 0.0, 0.0);
	glVertex3f(minX, maxY, maxZ);	// 7

	if (cubeIndex & (1 << 3))
		glColor3f(r, g, b);
	else
		glColor3f(1.0, 0.0, 0.0);
	glVertex3f(minX, maxY, minZ);	// 3

	if (cubeIndex & (1 << 3))
		glColor3f(r, g, b);
	else
		glColor3f(1.0, 0.0, 0.0);
	glVertex3f(minX, maxY, minZ);	// 3

	if (cubeIndex & (1 << 2))
		glColor3f(r, g, b);
	else
		glColor3f(1.0, 0.0, 0.0);
	glVertex3f(maxX, maxY, minZ);	// 2

	if (cubeIndex & (1 << 6))
		glColor3f(r, g, b);
	else
		glColor3f(1.0, 0.0, 0.0);
	glVertex3f(maxX, maxY, maxZ);	// 6

	if (cubeIndex & (1 << 7))
		glColor3f(r, g, b);
	else
		glColor3f(1.0, 0.0, 0.0);
	glVertex3f(minX, maxY, maxZ);	// 7


	glVertex3f(minX, maxY, maxZ);	// 7
	if (cubeIndex & (1 << 4))
		glColor3f(r, g, b);
	else
		glColor3f(1.0, 0.0, 0.0);
	glVertex3f(minX, minY, maxZ);	// 4

	if (cubeIndex & (1 << 5))
		glColor3f(r, g, b);
	else
		glColor3f(1.0, 0.0, 0.0);
	glVertex3f(maxX, minY, maxZ);	// 5

	if (cubeIndex & (1 << 6))
		glColor3f(r, g, b);
	else
		glColor3f(1.0, 0.0, 0.0);
	glVertex3f(maxX, maxY, maxZ);	// 6

	glVertex3f(maxX, maxY, maxZ);	// 6
	if (cubeIndex & (1 << 2))
		glColor3f(r, g, b);
	else
		glColor3f(1.0, 0.0, 0.0);
	glVertex3f(maxX, maxY, minZ);	// 2
	if (cubeIndex & (1 << 1))
		glColor3f(r, g, b);
	else
		glColor3f(1.0, 0.0, 0.0);
	glVertex3f(maxX, minY, minZ);	// 1

	if (cubeIndex & (1 << 5))
		glColor3f(r, g, b);
	else
		glColor3f(1.0, 0.0, 0.0);
	glVertex3f(maxX, minY, maxZ);	// 5

	glVertex3f(maxX, minY, maxZ);	// 5
	if (cubeIndex & (1 << 4))
		glColor3f(r, g, b);
	else
		glColor3f(1.0, 0.0, 0.0);
	glVertex3f(minX, minY, maxZ);	// 4
	if (cubeIndex & (1 << 0))
		glColor3f(r, g, b);
	else
		glColor3f(1.0, 0.0, 0.0);
	glVertex3f(minX, minY, minZ);	// 0
	if (cubeIndex & (1 << 1))
		glColor3f(r, g, b);
	else
		glColor3f(1.0, 0.0, 0.0);
	glVertex3f(maxX, minY, minZ);	// 1

	glEnd();
}

void drawBoundingBox()
{
	glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
	drawCube(0, g_gridMinXcoord, g_gridMaxXcoord, g_gridMinYcoord, g_gridMaxYcoord, g_gridMinZcoord, g_gridMaxZcoord, 1.0, 0.0, 0.0);
	//drawCube(0, msf.pmin.x, msf.pmax.x, msf.pmin.y, msf.pmax.y, msf.pmin.z, msf.pmax.z, 1.0, 0.0, 0.0);

}

void getMinMaxCoordinatesOfGrid()
{
	// There are no spheres, no need to calculate.
	if (pp.numproseq <= 0)
	{
		return;
	}

	// Set AtomMinXcoord/AtomMaxXcoord to x coordinate of center of the first sphere,
	// then let it compare to other sphere in for loop, finally get the min/max 
	// x coordinate among all spheres. Same rule apply to y/z axis.
	double AtomMinXcoord, AtomMinYcoord, AtomMinZcoord;
	double AtomMaxXcoord, AtomMaxYcoord, AtomMaxZcoord;
	double AtomMaxRadius;
	const atom firstSphere = pp.proseq[0];
	AtomMinXcoord = AtomMaxXcoord = firstSphere.x;
	AtomMinYcoord = AtomMaxYcoord = firstSphere.y;
	AtomMinZcoord = AtomMaxZcoord = firstSphere.z;
	AtomMaxRadius = firstSphere.radius;

	for (int i = 0; i < pp.numproseq; ++i)
	{
		// get min x
		if (pp.proseq[i].x < AtomMinXcoord)
		{
			AtomMinXcoord = pp.proseq[i].x;
		}
		// get max x
		else if (pp.proseq[i].x > AtomMaxXcoord)
		{
			AtomMaxXcoord = pp.proseq[i].x;
		}

		// get min y
		if (pp.proseq[i].y < AtomMinYcoord)
		{
			AtomMinYcoord = pp.proseq[i].y;
		}
		// get max y
		else if (pp.proseq[i].y > AtomMaxYcoord)
		{
			AtomMaxYcoord = pp.proseq[i].y;
		}

		// get min z
		if (pp.proseq[i].z < AtomMinZcoord)
		{
			AtomMinZcoord = pp.proseq[i].z;
		}
		// get max z
		else if (pp.proseq[i].z > AtomMaxZcoord)
		{
			AtomMaxZcoord = pp.proseq[i].z;
		}

		// get max radius
		if (pp.proseq[i].radius > AtomMaxRadius)
		{
			AtomMaxRadius = pp.proseq[i].radius;
		}

		//AtomMaxRadius = g_dblradius;

	}

	// Recenter molecular
	double xOffset = (AtomMaxXcoord + AtomMinXcoord) / 2;
	double yOffset = (AtomMaxYcoord + AtomMinYcoord) / 2;
	double zOffset = (AtomMaxZcoord + AtomMinZcoord) / 2;
	for (int i = 0; i < pp.numproseq; i++)
	{
		pp.proseq[i].x -= xOffset;
		pp.proseq[i].y -= yOffset;
		pp.proseq[i].z -= zOffset;
	}
	// Apply a delta for making sure the bounding of grid wrap all the atoms.
	double delta = 0.1;
	g_gridMinXcoord = AtomMinXcoord - xOffset - AtomMaxRadius - delta;
	g_gridMaxXcoord = AtomMaxXcoord - xOffset + AtomMaxRadius + delta;
	g_gridMinYcoord = AtomMinYcoord - yOffset - AtomMaxRadius - delta;
	g_gridMaxYcoord = AtomMaxYcoord - yOffset + AtomMaxRadius + delta;
	g_gridMinZcoord = AtomMinZcoord - zOffset - AtomMaxRadius - delta;
	g_gridMaxZcoord = AtomMaxZcoord - zOffset + AtomMaxRadius + delta;
}

void drawVoxel(const faceinfo& face, float sx, float sy, float sz)
{
	//Draw the triangles that were found.  There can be up to five per cube
	//glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
	//glBegin(GL_TRIANGLES);
	glColor3f(1.f, 1.f, 0.f);
	GLfloat x, y, z;

	x = g_gridMinXcoord + msf.verts[face.a].x * sx;
	y = g_gridMinYcoord + msf.verts[face.a].y * sy;
	z = g_gridMinZcoord + msf.verts[face.a].z * sz;
	glVertex3f(x, y, z);
	glColor3f(1.f, 1.f, 0.f);
	x = g_gridMinXcoord + msf.verts[face.b].x * sx;
	y = g_gridMinYcoord + msf.verts[face.b].y * sy;
	z = g_gridMinZcoord + msf.verts[face.b].z * sz;
	glVertex3f(x, y, z);
	glColor3f(1.f, 1.f, 0.f);
	x = g_gridMinXcoord + msf.verts[face.c].x * sx;
	y = g_gridMinYcoord + msf.verts[face.c].y * sy;
	z = g_gridMinZcoord + msf.verts[face.c].z * sz;
	glVertex3f(x, y, z);
	//glEnd();*/
}
__global__ void cudaDrawVoxel(faceinfo* facebuf, double* sx, double* sy, double* sz)
{
	int i = threadIdx.x;
	faceinfo vx = facebuf[i];
	printf("--------%d-----", i);
	//drawVoxel(vx, *sx, *sy, *sz);	
}


void drawTriangles()
{
	auto mode = g_showSurface ? GL_FILL : GL_LINE;
	glPolygonMode(GL_FRONT_AND_BACK, mode);
	glBegin(GL_TRIANGLES);
	glColor3f(1.0, 1.0, 0.0);

	double xScale = (g_gridMaxXcoord - g_gridMinXcoord) / msf.plength;
	double yScale = (g_gridMaxYcoord - g_gridMinYcoord) / msf.pwidth;
	double zScale = (g_gridMaxZcoord - g_gridMinZcoord) / msf.pheight;
	//msf.plength, msf.pwidth, msf.pheight, msf.scalefactor
#ifndef _CPU_
	for (int i = 0; i < msf.facenumber; ++i)
	{
		faceinfo vx = msf.faces[i];
		drawVoxel(vx, xScale, yScale, zScale);
	}
#else
	double *sx = 0, *sy = 0, *sz = 0;
	hipError_t cudaStatus;
	faceinfo *facebuf = 0;
	hipMalloc((void **)(&sx), sizeof(double));
	hipMalloc((void **)(&sy), sizeof(double));
	hipMalloc((void **)(&sz), sizeof(double));
	hipMalloc((void **)(&facebuf), sizeof(faceinfo)*msf.facenumber);
	
	hipMemcpy(sx, &xScale, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(sy, &yScale, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(sz, &zScale, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(facebuf, &msf.faces, sizeof(faceinfo)*msf.facenumber, hipMemcpyHostToDevice);

	cudaDrawVoxel << <1, 1 >> > (facebuf, sx, sy, sz);
	hipFree(facebuf);
	hipFree(sx);
	hipFree(sx);
	hipFree(sx);
#endif
	glEnd();
}


void display()
{
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	//glClear(GL_COLOR_BUFFER_BIT);   // Clear the color buffer with current clearing color

	glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
	glLoadIdentity();

	glTranslatef(g_x, g_y, g_z);
	glRotatef(g_rotate[0], 1.0, 0.0, 0.0);
	glRotatef(g_rotate[1], 0.0, 1.0, 0.0);
	glRotatef(g_rotate[2], 0.0, 0.0, 1.0);

	if (g_showAtoms)
		drawAtom();

	//drawAxis();
	if (g_showBoundingBox)
		drawBoundingBox();

	//if (g_showCuboids)
		//drawCuboids();

	if (g_showTriangles)
		drawTriangles();


	if (g_showPoints)
		drawSamplePoints();

	glFlush();  // Render now
}
// Reshape() function    
void Reshape(int w, int h)
{
	//adjusts the pixel rectangle for drawing to be the entire new window    
	glViewport(0, 0, (GLsizei)w, (GLsizei)h);
	//matrix specifies the projection transformation    
	glMatrixMode(GL_PROJECTION);
	// load the identity of matrix by clearing it.    
	glLoadIdentity();

	gluPerspective(g_numberOfGrid, (GLfloat)w / (GLfloat)h, 1.0, 200.0);
	//matrix specifies the modelview transformation    
	glMatrixMode(GL_MODELVIEW);
	// again  load the identity of matrix    
	glLoadIdentity();
}

void drawSphare(GLfloat x, GLfloat y, GLfloat z, GLfloat radius)
{
	glColor3f(1.0f, 0.0f, 1.0f);
	GLfloat alpha, beta; // Storage for coordinates and angles        
	int gradation = 10; //Number Of trangulation Trangulation from Sphare
	float X = x, Y = y, Z = z;
	for (alpha = 0.0; alpha < PI; alpha += PI / gradation)
	{
		glBegin(GL_TRIANGLE_STRIP);
		for (beta = 0.0; beta < 2.01*PI; beta += PI / gradation)
		{
			X = x + radius * cos(beta)*sin(alpha);
			Y = y + radius * sin(beta)*sin(alpha);
			Z = z + radius * cos(alpha);
			glVertex3f(X, Y, Z);
			X = x + radius * cos(beta)*sin(alpha + PI / gradation);
			Y = y + radius * sin(beta)*sin(alpha + PI / gradation);
			Z = z + radius * cos(alpha + PI / gradation);
			glVertex3f(X, Y, Z);
		}
		glEnd();
	}
}

void drawAtom()
{
	glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);

	for (int i = 0; i < pp.numproseq; ++i)
	{
		atom vx = pp.proseq[i];
		drawSphare(vx.x, vx.y, vx.z, vx.radius);
	}
}

void mouseCallBack(int btn, int state, int x, int y)
{
	// Mouse left button or right button presses down may lead to rotation
	if (btn == GLUT_LEFT_BUTTON || btn == GLUT_RIGHT_BUTTON)
	{
		if (state == GLUT_DOWN)
		{
			g_mouseState = DOWN;

			// When left or right button is pressed down, store the difference of
			// current position and previous rotation.
			g_xDiff = x - g_rotate[1];
			g_yDiff = -y + g_rotate[0];
		}
		else
		{
			g_mouseState = UP;
		}
	}
	else if (btn == WHEEL_UP) // zoom in
	{
		g_z++;
		glutPostRedisplay();
	}
	else if (btn == WHEEL_DOWN) // zoom out
	{
		g_z--;
		glutPostRedisplay();
	}
}

// Handler for mouse moving event
void onMouseMove(int x, int y)
{
	// only when the mouse is pressed down do the rotation
	if (g_mouseState == DOWN)
	{
		// Use the difference to calculate the finnal rotation value along x, y axes
		// Note that when mouse moves left and right means rotation around Y axes.
		// Mouse moves up and down means rotation around X axes
		g_rotate[1] = x - g_xDiff;
		g_rotate[0] = y + g_yDiff;

		glutPostRedisplay();
	}
}

void lighting(bool enable)
{
	if (enable)
	{
		GLfloat mat_specular[] = { 1.0, 1.0, 1.0, 1.0 };
		GLfloat mat_shininess[] = { 50.0 };
		GLfloat light_position[] = { 30.0, 1.0, 0.0, 1.0 };
		glClearColor(0.0, 0.0, 0.0, 0.0);
		glShadeModel(GL_SMOOTH);

		glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
		glMaterialfv(GL_FRONT, GL_SHININESS, mat_shininess);
		glLightfv(GL_LIGHT0, GL_POSITION, light_position);

		glEnable(GL_LIGHTING);
		glEnable(GL_LIGHT0);
		glEnable(GL_DEPTH_TEST);
	}
	else
	{
		glClearColor(1.0f, 1.0f, 1.0f, 1.0f);
		glDisable(GL_COLOR_MATERIAL);
		glDisable(GL_LIGHTING);
	}
}


void keyboardCallback(unsigned char key, int, int)
{
	switch (key)
	{
		// increase the number of grid
	case '+':
		g_numberOfGrid += 10;

		//marchingCube();
		//msf.marchingcube(g_nsurfMode);
		glutPostRedisplay();
		break;
		// decrease the number of grid
	case '-':
		if (g_numberOfGrid > 20)
		{
			g_numberOfGrid -= 10;
			//msf.marchingcube(g_nsurfMode);
			glutPostRedisplay();
		}
		else
		{
			std::cout << "Can not decrease the number of grid any more" << std::endl;
		}
		break;

		// increase the ios surface value
	case 'a':
		g_iosSurfaceValue *= 1.1;
		//msf.marchingcube(g_nsurfMode);
		glutPostRedisplay();
		break;
		// decrease the ios surface value

	case 'm':
		g_iosSurfaceValue *= 0.9;
		//msf.marchingcube(g_nsurfMode);
		glutPostRedisplay();
		break;

		// toggle display of bouding box of grid
	case 'b':
	case 'B':
		g_showBoundingBox = !g_showBoundingBox;
		glutPostRedisplay();
		break;

		// toggle display of atoms
	case 'd':
	case 'D':
		g_showAtoms = !g_showAtoms;
		glutPostRedisplay();
		break;

		// toggle display of random sample points
	case 'p':
	case 'P':
		g_showPoints = !g_showPoints;
		glutPostRedisplay();
		break;

		// toggle display of triangles
	case 't':
	case 'T':
		g_showTriangles = !g_showTriangles;
		glutPostRedisplay();
		break;

		// toggle display of cuboids
	case 'c':
	case 'C':
		g_showCuboids = !g_showCuboids;
		glutPostRedisplay();
		break;

		// toggle display of surface
	case 's':
	case 'S':
		g_showSurface = !g_showSurface;
		glutPostRedisplay();
		break;

		// toggle display of lighting
	case 'l':
	case 'L':
		g_showLight = !g_showLight;
		lighting(g_showLight);
		glutPostRedisplay();
		break;
	case 'u':
	case 'U':
		//setCUDA();
		break;
	default:
		break;
	}
}

void loadPdbfile(char* filename) {

	pp.loadpdb(filename, 2);
	pp.extractbb(0, -1, 1);

	getMinMaxCoordinatesOfGrid();
}

GLvoid initCamera()
{
	// Set up the perspective matrix.
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	// FOV, AspectRatio, NearClip, FarClip
	gluPerspective(60.0f, (float)(screen_width) / screen_height, 1.0f, 1000.0f);

	// Set up the camera matrices.
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
}
GLvoid initTexture()
{
	glEnable(GL_TEXTURE_2D);
}

GLvoid initLights()
{
	// Define each color component.
	GLfloat ambient[] = { 0.2f, 0.2f, 0.2f, 1.0f };
	GLfloat diffuse[] = { 0.7f, 0.7f, 0.7f, 1.0f };
	GLfloat specular[] = { 1.0f, 1.0f, 1.0f, 1.0f };

	// Set each color component.
	glLightfv(GL_LIGHT0, GL_AMBIENT, ambient);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, diffuse);
	glLightfv(GL_LIGHT0, GL_SPECULAR, specular);

	// Define and set position.
	float lightPos[4] = { 0, 0, 20, 1 };
	glLightfv(GL_LIGHT0, GL_POSITION, lightPos);

	// Turn on lighting.
	glEnable(GL_LIGHT0);
	glEnable(GL_LIGHTING);
}
GLvoid initMaterial()
{
	glColorMaterial(GL_FRONT_AND_BACK, GL_AMBIENT_AND_DIFFUSE);
	glEnable(GL_COLOR_MATERIAL);
}

GLvoid initColors()
{
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glColor3f(0.0, 0.0, 0.0);
	glLineWidth(1.0);
	glPointSize(5.0);
}

// Sets up OpenGL state.
GLvoid initGL()
{
	// Shading method: GL_SMOOTH or GL_FLAT
	glShadeModel(GL_SMOOTH);
	// Enable depth-buffer test.
	glEnable(GL_DEPTH_TEST);

	// Set the type of depth test.
	glDepthFunc(GL_LEQUAL);

	// 0 is near, 1 is far
	glClearDepth(1.0f);

	// Set camera settings.
	initCamera();

	// Set texture settings.
	initTexture();

	// Set lighting settings.
	initLights();

	// Set material settings.
	initMaterial();

	// Set color settings.
	initColors();
}

void calcuate_model() {
	
	if (g_nsurfMode == 1)
		msf.marchingcubeorigin2(2);
	else if (g_nsurfMode == 2)
		msf.marchingcube(2);

	clock_t remcfinish = clock();
	double duration = (double)(remcfinish - remcstart) / CLOCKS_PER_SEC;
	printf("Total time %.3f seconds\n", duration);
	//additional functions below
	msf.checkEuler();
	msf.computenorm();
	printf("No. vertices %d, No. triangles %d\n", msf.vertnumber, msf.facenumber);
	msf.calcareavolume();
	printf("Total area %.3f and volume %.3f\n", msf.sarea, msf.svolume);
	printf("Distinguish inner and outer surfaces\n");
	msf.surfaceinterior();
	printf("Calculate cavity number...\n");
	msf.cavitynumbers();
	printf("Cavity number is %d\n", msf.ncav);
	printf("Calculate cavity area and volume...\n");
	msf.cavitiesareavolume();
	printf("Cavity area %.3f and volume %.3f\n", msf.carea, msf.cvolume);
	printf("Calculate inner and outer atoms\n");
	msf.atomsinout(pp.promod[0].procha[0].chainseg.init, pp.promod[0].procha[pp.promod[0].nchain].chainseg.term, pp.proseq);
	msf.laplaciansmooth(1);
	msf.computenorm();
	msf.checkinoutpropa();

#ifdef _OUT_
	printf("Output 3D model\n");

	sprintf(filename, "%s.ply", outpname);
	msf.outputply(filename, pp.proseq, g_ncolor - 1, g_ninout - 1);
#endif

}
void opengl_run(int argc, char** argv) {
	glutInit(&argc, argv);          // Initialize GLUT
	//glutInitWindowSize(width, height);
	glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB | GLUT_DEPTH | GLUT_ALPHA);
	//glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE | GLUT_ALPHA | GLUT_DEPTH | GLUT_STENCIL);

	glutInitWindowSize(screen_width, screen_height);
	//glutInitWindowPosition(800, 600);
	glutCreateWindow("Molecular Surface SES");  // Create window with the given title

	// Initialize the scene.
	initGL();

	glutReshapeFunc(Reshape);
	glutDisplayFunc(display);       // Register callback handler for window re-paint event
	glutMouseFunc(mouseCallBack);
	glutKeyboardFunc(keyboardCallback);
	glutMotionFunc(onMouseMove);
	glutMainLoop();
}
void init_cuda(int argc, char** argv) {
	int dev = findCudaDevice(argc, (const char **)argv);
	printf("cuda device = %d", dev);

	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	printf("%s Global Memory Size %d GM", deviceProp.name, deviceProp.totalGlobalMem / 1024 / 1024 / 1024);
	hipSetDevice(dev);
}
int main(int argc, char** argv)
{
	init_cuda(argc, argv);
	char pdbfile[200] = "E:/WORK/2019_08_24/PDB_files/1bk2.pdb";
	char filename[200];

	bool bcolor;
	int i;
	clock_t remcstart, remcfinish;

	loadPdbfile(pdbfile);
	calcuate_model();

	opengl_run(argc, argv);

	return 1;
}

